#include "hip/hip_runtime.h"
#include "libwb/wb.h"

#define wbCheck(stmt)                                                     \
    do {                                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
            return -1;                                                          \
    }                                                                     \
    } while (0)

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

int main(int argc, char **argv) {
    wbArg_t args;
    int inputLength;
    float *hostInput1;
    float *hostInput2;
    float *hostOutput;
    float *deviceInput1;
    float *deviceInput2;
    float *deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 =
        (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 =
        (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *)malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

    int vectorSize = inputLength*sizeof(float);
    wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc((void **)&deviceInput1, vectorSize);
    hipMalloc((void **)&deviceInput2, vectorSize);
    hipMalloc((void **)&deviceOutput, vectorSize);
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    hipMemcpy(deviceInput1, hostInput1, vectorSize,hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, vectorSize,hipMemcpyHostToDevice);
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    dim3 GridDim((inputLength-1)/256+1,1,1);
    dim3 BlockDim(256,1,1);

    wbTime_start(Compute, "Performing CUDA computation");
    vecAdd<<<GridDim, BlockDim>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(hipMemcpy(hostOutput, deviceOutput, vectorSize, hipMemcpyDeviceToHost));

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}
